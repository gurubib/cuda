#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdlib>
#include <iostream>

#include <GL/glew.h>
#include <GL/freeglut.h>

#include "cutil_math.h"

__device__
float4 cross(float4 a, float4 b) {
	float3 res =cross(make_float3(a.x, a.y, a.z), make_float3(b.x, b.y, b.z));
	return(make_float4(res.x, res.y, res.z, 0.0f));
}

__device__
float fract(float x, float* iptr) {
	*iptr =  x - floorf(x);
	return x - floorf(x);
}

__device__
float2 mix(float2 x, float2 y, float a) {
	return x + (y - x) * a;
}

__device__
float4 mix(float4 x, float4 y, float a) {
	return x + (y - x) * a;
}

/* -*- mode: c++ -*- */

__constant__ float dt = 0.1f;

__global__
void resetSimulation(const int gridResolution,
	 float2* velocityBuffer,
	 float* pressureBuffer,
	 float4* densityBuffer) {
	
	int2 id = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);

	if (id.x < gridResolution && id.y < gridResolution) {
		velocityBuffer[id.x + id.y * gridResolution] = make_float2(0.0f, 0.0f);
		pressureBuffer[id.x + id.y * gridResolution] = 0.0f;
		densityBuffer[id.x + id.y * gridResolution] = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	}
}

// bilinear interpolation
__device__
float2 getBil(float2 p, int gridResolution,  float2* buffer) {
	p = clamp(p, make_float2(0.0f), make_float2(gridResolution));

	float2 p00 = buffer[(int)(p.x) + (int)(p.y) * gridResolution];
	float2 p10 = buffer[(int)(p.x) + 1 + (int)(p.y) * gridResolution];
	float2 p11 = buffer[(int)(p.x) + 1 + (int)(p.y + 1.0f) * gridResolution];
	float2 p01 = buffer[(int)(p.x) + (int)(p.y + 1.0f) * gridResolution];

	float flr;
	float t0 = fract(p.x, &flr);
	float t1 = fract(p.y, &flr);

	float2 v0 = mix(p00, p10, t0);
	float2 v1 = mix(p01, p11, t0);

	return mix(v0, v1, t1);
}

__device__
float4 getBil4(float2 p, int gridResolution,  float4* buffer) {
	p = clamp(p, make_float2(0.0f), make_float2(gridResolution));

	float4 p00 = buffer[(int)(p.x) + (int)(p.y) * gridResolution];
	float4 p10 = buffer[(int)(p.x) + 1 + (int)(p.y) * gridResolution];
	float4 p11 = buffer[(int)(p.x) + 1 + (int)(p.y + 1.0f) * gridResolution];
	float4 p01 = buffer[(int)(p.x) + (int)(p.y + 1.0f) * gridResolution];

	float flr;
	float t0 = fract(p.x, &flr);
	float t1 = fract(p.y, &flr);

	float4 v0 = mix(p00, p10, t0);
	float4 v1 = mix(p01, p11, t0);

	return mix(v0, v1, t1);
}

__global__
void advection(const int gridResolution,
	 float2* inputVelocityBuffer,
	 float2* outputVelocityBuffer) {
	int2 id = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);

	if (id.x > 0 && id.x < gridResolution - 1 &&
		id.y > 0 && id.y < gridResolution - 1) {
		float2 velocity = inputVelocityBuffer[id.x + id.y * gridResolution];

		float2 p = make_float2((float)id.x - dt * velocity.x, (float)id.y - dt * velocity.y);

		outputVelocityBuffer[id.x + id.y * gridResolution] = getBil(p, gridResolution, inputVelocityBuffer);
	}
	else {
		if (id.x == 0) outputVelocityBuffer[id.x + id.y * gridResolution] = -inputVelocityBuffer[id.x + 1 + id.y * gridResolution];
		if (id.x == gridResolution - 1) outputVelocityBuffer[id.x + id.y * gridResolution] = -inputVelocityBuffer[id.x - 1 + id.y * gridResolution];
		if (id.y == 0) outputVelocityBuffer[id.x + id.y * gridResolution] = -inputVelocityBuffer[id.x + 1 + (id.y + 1) * gridResolution];
		if (id.y == gridResolution - 1) outputVelocityBuffer[id.x + id.y * gridResolution] = -inputVelocityBuffer[id.x + 1 + (id.y - 1) * gridResolution];
	}
}

__global__
void advectionDensity(const int gridResolution,
	 float2* velocityBuffer,
	 float4* inputDensityBuffer,
	 float4* outputDensityBuffer) {
	int2 id = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);

	if (id.x > 0 && id.x < gridResolution - 1 &&
		id.y > 0 && id.y < gridResolution - 1) {
		float2 velocity = velocityBuffer[id.x + id.y * gridResolution];

		float2 p = make_float2((float)id.x - dt * velocity.x, (float)id.y - dt * velocity.y);

		outputDensityBuffer[id.x + id.y * gridResolution] = getBil4(p, gridResolution, inputDensityBuffer);
	}
	else {
		outputDensityBuffer[id.x + id.y * gridResolution] = make_float4(0.0f);
	}
}

__global__
void diffusion(const int gridResolution,
	 float2* inputVelocityBuffer,
	 float2* outputVelocityBuffer) {
	int2 id = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);

	float viscousity = 0.01f;
	float alpha = 1.0f / (viscousity * dt);
	float beta = 1.0f / (4.0f + alpha);

	if (id.x > 0 && id.x < gridResolution - 1 &&
		id.y > 0 && id.y < gridResolution - 1) {
		float2 vL = inputVelocityBuffer[id.x - 1 + id.y * gridResolution];
		float2 vR = inputVelocityBuffer[id.x + 1 + id.y * gridResolution];
		float2 vB = inputVelocityBuffer[id.x + (id.y - 1) * gridResolution];
		float2 vT = inputVelocityBuffer[id.x + (id.y + 1) * gridResolution];

		float2 velocity = inputVelocityBuffer[id.x + id.y * gridResolution];

		outputVelocityBuffer[id.x + id.y * gridResolution] = (vL + vR + vB + vT + alpha * velocity) * beta;
	}
	else {
		outputVelocityBuffer[id.x + id.y * gridResolution] = inputVelocityBuffer[id.x + id.y * gridResolution];
	}
}

__global__
void vorticity(const int gridResolution,  float2* velocityBuffer,
	 float* vorticityBuffer) {
	int2 id = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);

	if (id.x > 0 && id.x < gridResolution - 1 &&
		id.y > 0 && id.y < gridResolution - 1) {
		float2 vL = velocityBuffer[id.x - 1 + id.y * gridResolution];
		float2 vR = velocityBuffer[id.x + 1 + id.y * gridResolution];
		float2 vB = velocityBuffer[id.x + (id.y - 1) * gridResolution];
		float2 vT = velocityBuffer[id.x + (id.y + 1) * gridResolution];

		vorticityBuffer[id.x + id.y * gridResolution] = (vR.y - vL.y) - (vT.x - vB.x);
	}
	else {
		vorticityBuffer[id.x + id.y * gridResolution] = 0.0f;
	}
}

__global__
void addVorticity(const int gridResolution,  float* vorticityBuffer,
	 float2* velocityBuffer) {
	int2 id = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);

	const float scale = 0.2f;

	if (id.x > 0 && id.x < gridResolution - 1 &&
		id.y > 0 && id.y < gridResolution - 1) {
		float vL = vorticityBuffer[id.x - 1 + id.y * gridResolution];
		float vR = vorticityBuffer[id.x + 1 + id.y * gridResolution];
		float vB = vorticityBuffer[id.x + (id.y - 1) * gridResolution];
		float vT = vorticityBuffer[id.x + (id.y + 1) * gridResolution];

		float4 gradV = make_float4(vR - vL, vT - vB, 0.0f, 0.0f);
		float4 z = make_float4(0.0f, 0.0f, 1.0f, 0.0f);

		if (dot(gradV, gradV)) {
			float4 vorticityForce = scale * cross(gradV, z);
			velocityBuffer[id.x + id.y * gridResolution] += make_float2(vorticityForce.x, vorticityForce.y) * dt;
		}
	}
}

__global__
void divergence(const int gridResolution,  float2* velocityBuffer,
	 float* divergenceBuffer) {
	int2 id = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);

	if (id.x > 0 && id.x < gridResolution - 1 &&
		id.y > 0 && id.y < gridResolution - 1) {
		float2 vL = velocityBuffer[id.x - 1 + id.y * gridResolution];
		float2 vR = velocityBuffer[id.x + 1 + id.y * gridResolution];
		float2 vB = velocityBuffer[id.x + (id.y - 1) * gridResolution];
		float2 vT = velocityBuffer[id.x + (id.y + 1) * gridResolution];

		divergenceBuffer[id.x + id.y * gridResolution] = 0.5f * ((vR.x - vL.x) + (vT.y - vB.y));
	}
	else {
		divergenceBuffer[id.x + id.y * gridResolution] = 0.0f;
	}
}

__global__
void pressureJacobi(const int gridResolution,
	 float* inputPressureBuffer,
	 float* outputPressureBuffer,
	 float* divergenceBuffer) {
	int2 id = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);

	if (id.x > 0 && id.x < gridResolution - 1 &&
		id.y > 0 && id.y < gridResolution - 1) {

		float alpha = -1.0f;
		float beta = 0.25f;

		float vL = inputPressureBuffer[id.x - 1 + id.y * gridResolution];
		float vR = inputPressureBuffer[id.x + 1 + id.y * gridResolution];
		float vB = inputPressureBuffer[id.x + (id.y - 1) * gridResolution];
		float vT = inputPressureBuffer[id.x + (id.y + 1) * gridResolution];

		float divergence = divergenceBuffer[id.x + id.y * gridResolution];

		outputPressureBuffer[id.x + id.y * gridResolution] = (vL + vR + vB + vT + alpha * divergence) * beta;
	}
	else {
		if (id.x == 0) outputPressureBuffer[id.x + id.y * gridResolution] = inputPressureBuffer[id.x + 1 + id.y * gridResolution];
		if (id.x == gridResolution - 1) outputPressureBuffer[id.x + id.y * gridResolution] = inputPressureBuffer[id.x - 1 + id.y * gridResolution];
		if (id.y == 0) outputPressureBuffer[id.x + id.y * gridResolution] = inputPressureBuffer[id.x + (id.y + 1) * gridResolution];
		if (id.y == gridResolution - 1) outputPressureBuffer[id.x + id.y * gridResolution] = inputPressureBuffer[id.x + (id.y - 1) * gridResolution];
	}
}

__global__
void projection(const int gridResolution,
	 float2* inputVelocityBuffer,
	 float* pressureBuffer,
	 float2* outputVelocityBuffer) {
	int2 id = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);

	if (id.x > 0 && id.x < gridResolution - 1 &&
		id.y > 0 && id.y < gridResolution - 1) {
		float pL = pressureBuffer[id.x - 1 + id.y * gridResolution];
		float pR = pressureBuffer[id.x + 1 + id.y * gridResolution];
		float pB = pressureBuffer[id.x + (id.y - 1) * gridResolution];
		float pT = pressureBuffer[id.x + (id.y + 1) * gridResolution];

		float2 velocity = inputVelocityBuffer[id.x + id.y * gridResolution];

		outputVelocityBuffer[id.x + id.y * gridResolution] = velocity -  /* 0.5f **//* (1.0f / 256.0f) **/ make_float2(pR - pL, pT - pB);
	}
	else {
		if (id.x == 0) outputVelocityBuffer[id.x + id.y * gridResolution] = -inputVelocityBuffer[id.x + 1 + id.y * gridResolution];
		if (id.x == gridResolution - 1) outputVelocityBuffer[id.x + id.y * gridResolution] = -inputVelocityBuffer[id.x - 1 + id.y * gridResolution];
		if (id.y == 0) outputVelocityBuffer[id.x + id.y * gridResolution] = -inputVelocityBuffer[id.x + 1 + (id.y + 1) * gridResolution];
		if (id.y == gridResolution - 1) outputVelocityBuffer[id.x + id.y * gridResolution] = -inputVelocityBuffer[id.x + 1 + (id.y - 1) * gridResolution];
	}
}

__global__
void addForce(const float x, const float y, const float2 force,
	const int gridResolution,  float2* velocityBuffer,
	const float4 density,  float4* densityBuffer) {
	int2 id = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);

	float dx = ((float)id.x / (float)gridResolution) - x;
	float dy = ((float)id.y / (float)gridResolution) - y;

	float radius = 0.001f;

	float c = exp(-(dx * dx + dy * dy) / radius) * dt;

	velocityBuffer[id.x + id.y * gridResolution] += c * force;
	densityBuffer[id.x + id.y * gridResolution] += c * density;
}

// *************
// Visualization
// *************

__global__
void visualizationDensity(const int width, const int height,  float4* visualizationBuffer,
	const int gridResolution,  float4* densityBuffer) {
	int2 id = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);

	if (id.x < width && id.y < height) {
		float4 density = densityBuffer[id.x + id.y * width];
		visualizationBuffer[id.x + id.y * width] = density;
	}
}

__global__
void visualizationVelocity(const int width, const int height,  float4* visualizationBuffer,
	const int gridResolution,  float2* velocityBuffer) {
	int2 id = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);

	if (id.x < width && id.y < height) {
		float2 velocity = velocityBuffer[id.x + id.y * width];
		visualizationBuffer[id.x + id.y * width] = make_float4(((1.0f + velocity) / 2.0f).x, ((1.0f + velocity) / 2.0f).y, 0.0f, 0.0f);
	}
}

__global__
void visualizationPressure(const int width, const int height,  float4* visualizationBuffer,
	const int gridResolution,  float* pressureBuffer) {
	int2 id = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);

	if (id.x < width && id.y < height) {
		float pressure = pressureBuffer[id.x + id.y * width];
		visualizationBuffer[id.x + id.y * width] = make_float4((1.0f + pressure) / 2.0f);
	}
}

int gridResolution = 512;
int* d_gridResolution;

int inputVelocityBuffer = 0;
float2* d_velocityBuffer[2];

int inputDensityBuffer = 0;
float4* d_densityBuffer[2];
float4 densityColor;

int inputPressureBuffer = 0;
float* d_pressureBuffer[2];
float* d_divergenceBuffer;

float* d_vorticityBuffer;

size_t problemSize[2];

float2 force;

// visualization
int width = 512;
int height = 512;

float4* d_visualizationBufferGPU;
float4* visualizationBufferCPU;

int visualizationMethod = 0;

size_t visualizationSize[2];

void initSimulation() {
	// simulation
	problemSize[0] = gridResolution;
	problemSize[1] = gridResolution;

//Allocating device memory
	//velocityBuffer
	hipMalloc((void**)&d_velocityBuffer[0], sizeof(float2)*gridResolution*gridResolution);
	hipMalloc((void**)&d_velocityBuffer[1], sizeof(float2)*gridResolution*gridResolution);
	
	//densityBuffer
	hipMalloc((void**)&d_densityBuffer[0], sizeof(float4)*gridResolution*gridResolution);
	hipMalloc((void**)&d_densityBuffer[1], sizeof(float4)*gridResolution*gridResolution);

	//pressureBuffer
	hipMalloc((void**)&d_pressureBuffer[0], sizeof(float)*gridResolution*gridResolution);
	hipMalloc((void**)&d_pressureBuffer[1], sizeof(float)*gridResolution*gridResolution);

	//divergenceBuffer
	hipMalloc((void**)&d_divergenceBuffer, sizeof(float)*gridResolution*gridResolution);

	//vorticityBuffer
	hipMalloc((void**)&d_vorticityBuffer, sizeof(float)*gridResolution*gridResolution);

	densityColor.x = densityColor.y = densityColor.z = densityColor.w = 1.0f;


	// visualization
	visualizationSize[0] = width;
	visualizationSize[1] = height;

	//CPU
	visualizationBufferCPU = new float4[width * height];

	//GPU visualizationBuffer
	hipMalloc((void**)&d_visualizationBufferGPU, sizeof(float4)*width*height);
}

void resetSimulationHost() {
	resetSimulation <<<512, 512>>>(
		gridResolution,
		d_velocityBuffer[inputVelocityBuffer],
		d_pressureBuffer[inputPressureBuffer],
		d_densityBuffer[inputDensityBuffer]
		);
}

void resetPresssure() {
	resetSimulation <<<512, 512>>>(
		gridResolution,
		d_velocityBuffer[(inputVelocityBuffer + 1) % 2],
		d_pressureBuffer[inputPressureBuffer],
		d_densityBuffer[(inputVelocityBuffer + 1) % 2]
		);
}

void simulateAdvection() {
	advection<<<512, 512>>>(
		gridResolution,
		d_velocityBuffer[inputVelocityBuffer],
		d_velocityBuffer[(inputVelocityBuffer + 1) % 2]
		);
}


void simulateVorticity() {
	vorticity<<<512, 512>>>(
		gridResolution,
		d_velocityBuffer[inputVelocityBuffer],
		d_vorticityBuffer
		);

	addVorticity<<<512, 512>>>(
		gridResolution,
		d_vorticityBuffer,
		d_velocityBuffer[inputVelocityBuffer]
		);
}

void simulateDiffusion() {
	for (int i = 0; i < 10; ++i) {
		diffusion<<<512,512>>>(
			gridResolution,
			d_velocityBuffer[inputVelocityBuffer],
			d_velocityBuffer[(inputVelocityBuffer + 1) % 2]
			);

		inputVelocityBuffer = (inputVelocityBuffer + 1) % 2;
	}
}

void projection() {
	divergence<<<512,512>>>(
		gridResolution,
		d_velocityBuffer[inputVelocityBuffer],
		d_divergenceBuffer
		);

	resetPresssure();

	for (int i = 0; i < 10; ++i) {
		pressureJacobi<<<512,512>>>(
			gridResolution,
			d_pressureBuffer[inputPressureBuffer],
			d_pressureBuffer[(inputPressureBuffer + 1) % 2],
			d_divergenceBuffer
			);


		inputPressureBuffer = (inputPressureBuffer + 1) % 2;
	}

	projection<<<512,512>>>(
		gridResolution,
		d_velocityBuffer[inputVelocityBuffer],
		d_pressureBuffer[inputPressureBuffer],
		d_velocityBuffer[(inputVelocityBuffer + 1) % 2]
		);

	inputVelocityBuffer = (inputVelocityBuffer + 1) % 2;
}

void simulateDensityAdvection() {
	advectionDensity<<<512,512>>>(
		gridResolution,
		d_velocityBuffer[inputVelocityBuffer],
		d_densityBuffer[inputDensityBuffer],
		d_densityBuffer[(inputDensityBuffer + 1) % 2]
		);

	inputDensityBuffer = (inputDensityBuffer + 1) % 2;
}

void addForce(int x, int y, float2 force) {
	float fx = (float)x / width;
	float fy = (float)y / height;

	addForce<<<512,512>>>(
		fx,
		fy,
		force,
		gridResolution,
		d_velocityBuffer[inputVelocityBuffer],
		densityColor,
		d_densityBuffer[inputDensityBuffer]
		);
}

void simulationStep() {
	simulateAdvection();
	simulateDiffusion();
	simulateVorticity();
	projection();
	simulateDensityAdvection();
}

void visualizationStep() {
	switch (visualizationMethod) {
	case 0:
		visualizationDensity<<<512,512>>>(
			width,
			height,
			d_visualizationBufferGPU,
			gridResolution,
			d_densityBuffer[inputDensityBuffer]
			);
		break;
	case 1:
		visualizationVelocity<<<512,512 >>>(
			width,
			height,
			d_visualizationBufferGPU,
			gridResolution,
			d_velocityBuffer[inputVelocityBuffer]
			);
		break;
	case 2:
		visualizationPressure<<<512,512>>>(
			width,
			height,
			d_visualizationBufferGPU,
			gridResolution,
			d_pressureBuffer[inputPressureBuffer]
			);
		break;
	
	}

	hipMemcpy(visualizationBufferCPU, d_visualizationBufferGPU, sizeof(float4) * width * height, hipMemcpyDeviceToHost);
	glDrawPixels(width, height, GL_RGBA, GL_FLOAT, visualizationBufferCPU);
}

// OpenGL
int method = 1;
bool keysPressed[256];

void initOpenGL() {
	glewExperimental = GL_TRUE;
	GLenum err = glewInit();
	if (GLEW_OK != err) {
		std::cerr << "Error: " << glewGetErrorString(err) << std::endl;
	}
	else {
		if (GLEW_VERSION_3_0)
		{
			std::cout << "Driver supports OpenGL 3.0\nDetails:" << std::endl;
			std::cout << "  Using GLEW " << glewGetString(GLEW_VERSION) << std::endl;
			std::cout << "  Vendor: " << glGetString(GL_VENDOR) << std::endl;
			std::cout << "  Renderer: " << glGetString(GL_RENDERER) << std::endl;
			std::cout << "  Version: " << glGetString(GL_VERSION) << std::endl;
			std::cout << "  GLSL: " << glGetString(GL_SHADING_LANGUAGE_VERSION) << std::endl;
		}
	}

	glClearColor(0.17f, 0.4f, 0.6f, 1.0f);
}

void display() {
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glDisable(GL_DEPTH_TEST);

	simulationStep();
	visualizationStep();

	glEnable(GL_DEPTH_TEST);
	glutSwapBuffers();
}

void idle() {
	glutPostRedisplay();
}

void keyDown(unsigned char key, int x, int y) {
	keysPressed[key] = true;
}

void keyUp(unsigned char key, int x, int y) {
	keysPressed[key] = false;
	switch (key) {
	case 'r':
		resetSimulationHost();
		break;

	case 'd':
		visualizationMethod = 0;
		break;
	case 'v':
		visualizationMethod = 1;
		break;
	case 'p':
		visualizationMethod = 2;
		break;

	case '1':
		densityColor.x = densityColor.y = densityColor.z = densityColor.w = 1.0f;
		break;

	case '2':
		densityColor.x = 1.0f;
		densityColor.y = densityColor.z = densityColor.w = 0.0f;
		break;

	case '3':
		densityColor.y = 1.0f;
		densityColor.x = densityColor.z = densityColor.w = 0.0f;
		break;

	case '4':
		densityColor.z = 1.0f;
		densityColor.x = densityColor.y = densityColor.w = 0.0f;
		break;

	case 27:
		exit(0);
		break;
	}
}

int mX, mY;

void mouseClick(int button, int state, int x, int y) {
	if (button == GLUT_LEFT_BUTTON)
		if (state == GLUT_DOWN) {
			mX = x;
			mY = y;
		}
}

void mouseMove(int x, int y) {
	force.x = (float)(x - mX);
	force.y = -(float)(y - mY);
	//addForce(mX, height - mY, force);
	addForce(256, 256, force);
	mX = x;
	mY = y;
}

void reshape(int newWidth, int newHeight) {
	width = newWidth;
	height = newHeight;
	glViewport(0, 0, width, height);
}

int main(int argc, char* argv[]) {
	glutInit(&argc, argv);
	glutInitContextVersion(3, 0);
	glutInitContextFlags(GLUT_CORE_PROFILE | GLUT_DEBUG);
	glutInitDisplayMode(GLUT_RGBA | GLUT_DEPTH | GLUT_DOUBLE);
	glutInitWindowSize(width, height);
	glutCreateWindow("GPGPU 13. labor: Incompressible fluid simulation");

	initOpenGL();

	glutDisplayFunc(display);
	glutIdleFunc(idle);
	glutReshapeFunc(reshape);
	glutKeyboardFunc(keyDown);
	glutKeyboardUpFunc(keyUp);
	glutMouseFunc(mouseClick);
	glutMotionFunc(mouseMove);

	// OpenCL processing
	initSimulation();

	glutMainLoop();
	return(0);
}
